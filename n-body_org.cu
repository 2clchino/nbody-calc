
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#define N (3)
#define dt 0.001
#define Soften (1e-6)
// #define NT 256
// #define NB (N/NT)
#define Frame 10

//加速度の計算
__global__ void kernel(float *d_x,float *d_y,
		       float *d_z,float *d_vx, float *d_vy,
		       float *d_vz,float *d_ax, float *d_ay,
		       float *d_az,float *d_m){
  int i,j;
  float rx,ry,rz;
  float dist2, dist6, invDist3,s;
  i = blockDim.x*blockIdx.x+threadIdx.x;
  for(j=0;j<N;j++){
    rx=d_x[j]-d_x[i];
    ry=d_y[j]-d_y[i];
    rz=d_z[j]-d_z[i];
    // 2天体間の距離を計算
    dist2 = rx*rx + ry*ry + rz*rz + Soften;
    // m/r^3 の計算
    dist6 = dist2*dist2*dist2;
    invDist3 = 1.0/sqrt(dist6);
    s = d_m[j]*invDist3;
    //天体jによる加速度を加算
    d_ax[i] = d_ax[i] + rx*s;
    d_ay[i] = d_ay[i] + ry*s;
    d_az[i] = d_az[i] + rz*s;
  }
}

// tで積分
__global__ void integrate(float *d_x, float *d_y, float *d_z,
			  float *d_vx, float *d_vy, float *d_vz, float *d_ax, float *d_ay, float *d_az){
  int i=blockIdx.x*blockDim.x + threadIdx.x;
  d_x[i] = d_x[i] + dt*d_vx[i];
  d_y[i] = d_y[i] + dt*d_vy[i];
  d_z[i] = d_z[i] + dt*d_vz[i];
  d_vx[i] = d_vx[i] + dt*d_ax[i];
  d_vy[i] = d_vy[i] + dt*d_ay[i];
  d_vz[i] = d_vz[i] + dt*d_az[i];
}

void initial(float *x,float *y,float *z,float *vx,float *vy,float *vz,float *m){
  int i;
  //乱数で配置を決定
  //x,y座標，x,y方向速度が‐1~1の範囲に収まるように決定
  srand(N);
  for(i=0;i<N;i++){
    x[i] = (float)rand()/RAND_MAX*2.0 - 1.0;
    y[i] = (float)rand()/RAND_MAX*2.0 - 1.0;
    z[i] = (float)rand()/RAND_MAX*2.0 - 1.0;;
    m[i] = 1.0f;
    vx[i] = (float)rand()/RAND_MAX*2.0 - 1.0;
    vy[i] = (float)rand()/RAND_MAX*2.0 - 1.0;
    vz[i] = (float)rand()/RAND_MAX*2.0 - 1.0;;
  }
}

int main(void){
  //GPUのメモリ上に確保
  float *d_x,*d_y,*d_z,*d_m;
  float *d_vx,*d_vy,*d_vz;
  float *d_ax,*d_ay,*d_az;
  hipMallocManaged((void **)&d_x, (N*sizeof(float)));
  hipMallocManaged((void **)&d_y, (N*sizeof(float)));
  hipMallocManaged((void **)&d_z, (N*sizeof(float)));
  hipMallocManaged((void **)&d_m, (N*sizeof(float)));
  hipMallocManaged((void **)&d_vx, (N*sizeof(float)));
  hipMallocManaged((void **)&d_vy, (N*sizeof(float)));
  hipMallocManaged((void **)&d_vz, (N*sizeof(float)));
  hipMalloc((void **)&d_ax, (N*sizeof(float)));
  hipMalloc((void **)&d_ay, (N*sizeof(float)));
  hipMalloc((void **)&d_az, (N*sizeof(float)));
  initial(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_m);
  for(int t = 0; t < 10; t++){
    for (int i = 0; i < N; i++){
      printf("%lf %lf %lf\n", d_x[i], d_y[i], d_z[i]);
    }
    printf("\n\n");
    kernel<<<1,3>>>(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_ax, d_ay, d_az, d_m);
    integrate<<<1,3>>>(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_ax, d_ay, d_az);
    hipDeviceSynchronize();
  }
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_m);
  hipFree(d_vx);
  hipFree(d_vy);
  hipFree(d_vz);
  hipFree(d_ax);
  hipFree(d_ay);
  hipFree(d_az);
  hipDeviceReset();
  return 0;
}
